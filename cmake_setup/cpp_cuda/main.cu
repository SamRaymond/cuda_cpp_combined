#include "simpleAddCPU.hpp"
#include "simpleAddGPU.h"
#include <stdio.h>

int main(void){

    long N = 300*1000*1000; //~300 variables in Graphyt
    // int *a;
    // a = new int[N];
    // int *b;
    // b = new int[N];
    // int *c;
    // c = new int[N];

    float* a;
    hipMallocManaged(&a, N*sizeof(float));
    float* b;
    hipMallocManaged(&b, N*sizeof(float));
    float* c;
    hipMallocManaged(&c, N*sizeof(float));
    
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    randomize_cuda<<<1, blockSize>>>(a,b,c,N);
    hipDeviceSynchronize();
    double t1 = omp_get_wtime();
    add_cuda<<<numBlocks, blockSize>>>(a,b,c,N);
    hipDeviceSynchronize();
    double t2 = omp_get_wtime();
    
    for(int p=0;p<10;p++){
        printf("%f, %f,%f \n",a[p],b[p],c[p]);
    }
    printf("Particles: %li\nGPU Time: %.2fs\n",N/300,t2-t1);

    double t3 = omp_get_wtime();
    add_CPU(a,b,c,N);
    double t4 = omp_get_wtime();
    for(int p=0;p<10;p++){
        printf("%f, %f,%f \n",a[p],b[p],c[p]);
    }

    printf("Particles: %li\nGPU Time: %.2fs\nCPU Time: %.2fs\nApprox. Speed-up: %.1f\n",N/300,t2-t1,t4-t3,(t4-t3)/(t2-t1));
    // delete a;
    // delete b;
    // delete c;
    hipDeviceSynchronize();
    hipFree(a);
    hipFree(b);
    hipFree(c);
    return 0;
}
