#include "hip/hip_runtime.h"
#include "simpleAddGPU.h"
#include <stdio.h>
__global__ void randomize_cuda(double* a,double* b, double* c,long N){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(index<N){
        a[index] = index;
        b[index] = index/2;
        c[index] = 0;
    }
};

__global__ void add_cuda(double* a,double* b,double* c,long N){
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index == 2){
	printf("%f,%p\n",c[index],(void*)&c[index]);
}

    if(index<N){
        c[index] += a[index] + b[index];
        for(int q=0; q<100000; q++){
            c[index] +=0.1;
            c[index] -=0.05;
        }
    }
};

