#include <simpleAddCPU.hpp>
#include <simpleAddGPU.h>
#include <stdio.h>

int main(void){

    long N = 1*1000*1000;
    // int *a;
    // a = new int[N];
    // int *b;
    // b = new int[N];
    // int *c;
    // c = new int[N];

    double* a;
    hipMallocManaged(&a, N*sizeof(double));
    double* b;
    hipMallocManaged(&b, N*sizeof(double));
    double* c;
    hipMallocManaged(&c, N*sizeof(double));
    
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    randomize_cuda<<<1, blockSize>>>(a,b,c,N);
    hipDeviceSynchronize();
    add_cuda<<<numBlocks, blockSize>>>(a,b,c,N);
    hipDeviceSynchronize();
    // for(int p=0;p<10;p++){
    //     printf("%f, %f,%f \n",a[p],b[p],c[p]);
    // }
    add_CPU(a,b,c,N);
  //  for(int p=0;p<10;p++){
  //      printf("%f, %f,%f \n",a[p],b[p],c[p]);
  //  }
    // delete a;
    // delete b;
    // delete c;
    hipDeviceSynchronize();
    hipFree(a);
    hipFree(b);
    hipFree(c);
    return 0;
}
