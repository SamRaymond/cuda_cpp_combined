#include "hip/hip_runtime.h"
#include <simpleAddGPU.h>

__global__ void randomize_cuda(double* a,double* b, double* c,long N){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(index<N){
        a[index] = index;
        b[index] = index/2;
        c[index] = 0;
    }
};

__global__ void add_cuda(double* a,double* b,double* c,long N){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index<N){
        c[index] = a[index] + b[index];
        for(int q=0; q<50000; q++){
            c[index] = c[index]+1;
            c[index] = c[index]-1;
        }
    }
};

