#include "hip/hip_runtime.h"
#include <simpleAddCPU.hpp>

__global__ void randomize_cuda(double* a,double* b, double* c,long N){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(index<N){
        a[index] = index;
        b[index] = index/2;
        c[index] = 0;
    }
};

__global__ void add_cuda(double* a,double* b,double* c,long N){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index<N){
        c[index] = a[index] + b[index];
        for(int q=0; q<50000; q++){
            c[index] = c[index]+1;
            c[index] = c[index]-1;
        }
    }
};

int main(void){

    long N = 100*1000*1000;
    // int *a;
    // a = new int[N];
    // int *b;
    // b = new int[N];
    // int *c;
    // c = new int[N];

    double* a;
    hipMallocManaged(&a, N*sizeof(double));
    double* b;
    hipMallocManaged(&b, N*sizeof(double));
    double* c;
    hipMallocManaged(&c, N*sizeof(double));
    
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    randomize_cuda<<<1, blockSize>>>(a,b,c,N);
    hipDeviceSynchronize();
    add_cuda<<<numBlocks, blockSize>>>(a,b,c,N);
    hipDeviceSynchronize();
    for(int p=0;p<10;p++){
        printf("%f, %f,%f \n",a[p],b[p],c[p]);
    }
    add_CPU(a,b,c,N);
    for(int p=0;p<10;p++){
        printf("%f, %f,%f \n",a[p],b[p],c[p]);
    }
    // delete a;
    // delete b;
    // delete c;
    hipDeviceSynchronize();
    hipFree(a);
    hipFree(b);
    hipFree(c);
    return 0;
}

